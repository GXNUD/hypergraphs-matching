
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <math.h>
#include <time.h>
#include <stdio.h>

using namespace std;


__global__ void kernel_Sum(double *A, double *B, double *C, int N){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < N){
    C[i] = A[i] + B[i];
  }

}

int main(int argc, char *argv[])
{

    vector<double> a;
    double *ap = (double*)malloc((a.size())*sizeof(double));
    vector<double> b;
    double *bp = (double*)malloc((b.size())*sizeof(double));
    vector<double> c;
    double *cp = (double*)malloc((c.size())*sizeof(double));

    // declaración de variables cuda para la GPU
    double *d_Ap, *d_Bp, *d_Cp;
    
    a.push_back(999.25);
    a.push_back(888.50);
    a.push_back(777.25);

    b.push_back(999.25);
    b.push_back(888.50);
    b.push_back(777.25);
    c.push_back(0.0);
    c.push_back(0.0);
    c.push_back(0.0);

    int N = c.size();
    cout << N << '\n';

    hipMalloc((void **)&d_Ap , N*sizeof(double));
    hipMalloc((void **)&d_Bp , N*sizeof(double));
    hipMalloc((void **)&d_Cp , N*sizeof(double));

    //int threadsPerBlock = 512;
    //int blocksPerGrid =  ceil(double(N)/double(threadsPerBlock));
    //
    for(int i = 0; i< a.size(); i++){
      ap[i]= a[i];
      bp[i]= b[i];
      cp[i]= c[i];
    }

    for(int i = 0; i < c.size(); i++)
    {
      cp[i]=ap[i]+bp[i];
      cout << cp[i] << endl;
    }
    cout << "--FINALIZA SECUENCIAL------" << endl;

    hipMemcpy (d_Ap, ap , N*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy (d_Bp, bp , N*sizeof(double),hipMemcpyHostToDevice);

    kernel_Sum <<<1,1>>>(d_Ap, d_Bp, d_Cp, N);

    hipMemcpy (cp, d_Cp , N*sizeof(double),hipMemcpyHostToDevice);

    for(int i = 0; i< N; i++){
      cout << cp[i] << '\n';
    }

    cout << "---FINALIZA PARALELO-------" << endl;


    free(ap);
    free(cp);
    free(bp);
    hipFree(d_Ap);
    hipFree(d_Bp);
    hipFree(d_Cp);

    return EXIT_SUCCESS;
}
